#include "hip/hip_runtime.h"
﻿#include "lab3.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include ""
#define R 0
#define G 1
#define B 2
__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;
	if (yt < ht && xt < wt && mask[curt] > 127.0f) {
		const int yb = oy + yt, xb = ox + xt;
		const int curb = wb*yb + xb;
		if (0 <= yb && yb < hb && 0 <= xb && xb < wb) {
			output[curb * 3 + R] = target[curt * 3 + R];
			output[curb * 3 + G] = target[curt * 3 + G];
			output[curb * 3 + B] = target[curt * 3 + B];
		}
	}
}




__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;

	float targetNeiborUp, targetNeiborDown, targetNeiborLeft, targetNeiborRight, targetNeiborSum;
	float backgroundNeiborUp, backgroundNeiborLeft, backgroundNeiborDown, backgroundNeiborRight, backgroundSum;
	if (yt < ht && xt < wt) {
		const int yb = oy + yt, xb = ox + xt;
		const int curb = wb*yb + xb;

		for (int i = 0; i < 3; i++){

			backgroundSum = 0;
			targetNeiborSum = 0;
			// 判斷有沒有在範圍內
			//
			targetNeiborLeft = (curt - 1 < 0)? target[curt * 3 + i] : target[curt * 3 + i - 1 * 3];
			targetNeiborRight = (curt + 1 > wt*ht) ? target[curt * 3 + i] : target[curt * 3 + i + 1 * 3];
			targetNeiborUp = (curt - wt < 0) ? target[curt * 3 + i] : target[curt * 3 + i - wt * 3];
			targetNeiborDown = (curt + wt > wt*ht) ? target[curt * 3 + i] : target[curt * 3 + i + wt * 3];

			targetNeiborSum = targetNeiborLeft + targetNeiborRight + targetNeiborUp + targetNeiborDown;

			if (curt - 1 < 0)
			{
				backgroundNeiborLeft = background[curb * 3 + i - 1 * 3];
			}
			else
			{
				backgroundNeiborLeft = mask[curt - 1] > 127.0 ? 0 : background[curb * 3 + i - 1 * 3];
			}

			if (curt + 1 > wt*ht)
			{
				backgroundNeiborRight = background[curb * 3 + i + 1 * 3];
			}
			else
			{
				backgroundNeiborRight = mask[curt + 1] > 127.0 ? 0 : background[curb * 3 + i + 1 * 3];
			}

			if (curt - wt < 0)
			{
				backgroundNeiborUp = background[curb * 3 + i - wb * 3];
			}
			else
			{
				backgroundNeiborUp = mask[curt - wt] > 127.0 ? 0 : background[curb * 3 + i - wb * 3];

			}

			if (curt + wt > wt*ht)
			{
				backgroundNeiborDown = background[curb * 3 + i + wb * 3];
			}
			else
			{
				backgroundNeiborDown = mask[curt + wt] > 127.0 ? 0 : background[curb * 3 + i + wb * 3];
			}


			backgroundSum = backgroundNeiborLeft + backgroundNeiborRight + backgroundNeiborUp + backgroundNeiborDown;
			fixed[curt * 3 + i] = 4.0 * target[curt * 3 + i] - targetNeiborSum + backgroundSum;
		}
	}

}
__global__ void PoissonImageCloningIteration(
	const float *fixed,
	const float *mask,
	float *target,
	float *output,
	const int wt, const int ht
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;

	float OutputNeiborDown, OutputNeiborRight, OutputNeiborLeft, OutputNeiborTop;


	if (yt < ht && xt < wt) {
		//Run 
		for (int i = 0; i < 3; i++){

			float  count_neibor = 0.0f;

			if (curt - 1 > 0) // ¥ª
			{
				OutputNeiborLeft = mask[curt - 1] < 127.0f ? 0 : target[curt * 3 + i - 1 * 3];
				count_neibor += OutputNeiborLeft;
			}
			if (curt + 1 < wt*ht) // ¥k
			{
				OutputNeiborRight = (mask[curt + 1] < 127.0f) ? 0 : target[curt * 3 + i + 1 * 3];
				count_neibor += OutputNeiborRight;
			}
			if (curt - wt > 0) //¤W
			{
				OutputNeiborTop = (mask[curt - wt] < 127.0f) ? 0 : target[curt * 3 + i - wt * 3];
				count_neibor += OutputNeiborTop;
			}
			if (curt + wt < wt*ht) //¤U
			{
				OutputNeiborDown = (mask[curt + wt] < 127.0f) ? 0 : target[curt * 3 + i + wt * 3];
				count_neibor += OutputNeiborDown;
			}


			output[curt * 3 + i] = (1.0 / 4.0)*(fixed[curt * 3 + i] + count_neibor);

		}

	}
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf1, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf2, 3 * wt*ht*sizeof(float));

	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);

	CalculateFixed << <gdim, bdim >> >(
		background, target, mask, fixed,
		wb, hb, wt, ht, oy, ox
		);

	hipMemcpy(buf1, target, sizeof(float)* 3 * wt*ht, hipMemcpyDeviceToDevice);
	
	for (int i = 0; i < 10000; ++i)
	{
		PoissonImageCloningIteration << <gdim, bdim >> >(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningIteration << <gdim, bdim >> >(fixed, mask, buf2, buf1, wt, ht);
	}

	hipMemcpy(output, background, wb*hb*sizeof(float)* 3, hipMemcpyDeviceToDevice);
	
	
	SimpleClone << <gdim, bdim >> >(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
		);
	
	/*SimpleClone << <dim3(CeilDiv(wt, 32), CeilDiv(ht, 16)), dim3(32, 16) >> >(
		background, target, mask, output,
		wb, hb, wt, ht, oy, ox
		);*/

	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}
