#include "hip/hip_runtime.h"
#include "lab2.h"

// Macro functions - YUV and RGB converting.
#define clip(x) ((x) > 255 ? 255 : (x) < 0 ? 0 : (int)x)
#define RGBtoY(R, G, B) clip( ( 0.299 * R) + ( 0.587 * G) + ( 0.114 * B)      )
#define RGBtoU(R, G, B) clip( (-0.169 * R) + (-0.331 * G) + ( 0.500 * B) + 128)
#define RGBtoV(R, G, B) clip( ( 0.500 * R) + (-0.419 * G) + (-0.081 * B) + 128)

static const unsigned W = 1920;
static const unsigned H = 1080;
static const unsigned NFRAME = 240;

struct Vector3D
{
 unsigned char x, y, z;
 Vector3D(unsigned char x, unsigned char y, unsigned char z)
 {
  this->x = x;
  this->y = y;
  this->z = z;
 }
};

Lab2VideoInfo tmpInfo;
struct Lab2VideoGenerator::Impl {
 int t = 0;
};



Lab2VideoGenerator::Lab2VideoGenerator() : impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

__global__ void PCG2()
{


}
__device__ int* PCGRecurOne(int x, int y, int t, int part)
{

 //printf("x = %d , y = %d ,cosx = %f\n ",x,y,cosf(x));
 int windowDis = H*H + W*W;
 int dis = x*x + y*y;
 float disMod = (float)dis / (float)windowDis;
 //printf("dixmod =%f ,%d \n", (255 * disMod), (int)(255 * disMod));
 if (part == 1)
 {
  int tmpt = t < 10 ? t = 10 : t;

  float func = cosf(2*(y + t)) + cosf(x + t);
  if (func < 0.1 && func> -0.1)
  {
   int RGB[3] = { 255, 255 * disMod, };
   //printf("1RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
   return RGB;
  }

  else
  {
   int RGB[3] = { -1, -1, -1 };
   //printf("3RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
   return RGB;
  }
 }
 int RGB[3] = { -1, -1, -1 };
 //printf("4RGB = %d , %d , %d\n", RGB[0], RGB[1], RGB[2]);
 return RGB;
}
__global__ void SetColor(uint8_t *yuv, int x1, int y1, int vectorX, int vectorY, int * color)
{
 int t = blockIdx.x * blockDim.x + threadIdx.x;
 y1 += vectorY / abs(vectorX)*t;
 x1 += vectorX / abs(vectorX)*t;
 int idx = (y1)*W + x1;
 if (idx > W*H || x1<0 || x1>W || y1<0 || y1>H)return;
 int rowOfY = (idx / W);
 int columnOfY = (idx%W);
 int rowOfUV = rowOfY / 2;
 int columnOfUV = columnOfY / 2;
 int uvWidth = W / 2;
 int uvIdx = rowOfUV *uvWidth + columnOfUV;

 yuv[idx] = RGBtoY(color[0], color[1], color[2]);
 yuv[W*H + uvIdx] = RGBtoU(color[0], color[1], color[2]);
 yuv[W*H + W*H / 4 + uvIdx] = RGBtoV(color[0], color[1], color[2]);

}
__device__ void Line(uint8_t *yuv, float x1, float y1, float x2, float y2, int * color)
{
 int idx = (int)y1*W + x1;
 //if (idx>W*H || x1<0 || x2<0 || x1>W || x2>W || y1<0 || y2<0 || y1>H || y2>H)return;

 int rowOfY = (idx / W);
 int columnOfY = (idx%W);
 int rowOfUV = rowOfY / 2;
 int columnOfUV = columnOfY / 2;
 int uvWidth = W / 2;
 int uvIdx = rowOfUV *uvWidth + columnOfUV;
 float vectorX = x2 - x1;
 float vectorY = y2 - y1;
 //SetColor << <1, abs(vectorX) >> >(yuv,x1,y1, vectorX, vectorY, color);

 int dist = 0;
 int maxDis = H;

 for (int i = 0; i < abs(vectorX); i++)
 {
  if (idx>W*H || x1<0 || x2<0 || x1>W || x2>W || y1<0 || y2<0 || y1>H || y2>H)continue;

  dist = y1;


  yuv[idx] = RGBtoY(255 - 100 * dist / maxDis, 0, 0);
  yuv[W*H + uvIdx] = RGBtoU(255 - 100 * dist / maxDis, 0, 0);
  yuv[W*H + W*H / 4 + uvIdx] = RGBtoV(255 - 100 * dist / maxDis, 0, 0);


  y1 += vectorY / abs(vectorX);
  x1 += vectorX / abs(vectorX);
  idx = (int)(y1)*W + x1;
  rowOfY = (idx / W);
  columnOfY = (idx%W);
  rowOfUV = rowOfY / 2;
  columnOfUV = columnOfY / 2;
  uvWidth = W / 2;
  uvIdx = rowOfUV *uvWidth + columnOfUV;
 }
}

__global__ void PCGRecur(int beginX, int beginY, int x, int y, uint8_t *yuv, int t, int orginIdx, int recurId, int caseID){


 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 int locateIdx = y * W + x;
 int width = x;
 int height = y;


 int tPCG = t;
 int recurIdx = recurId;

 int origWidth = orginIdx%W;
 int origHeight = orginIdx / W;
 int uvWidth = width / 2;
 int uvHeight = height / 2;
 int origUVWidth = origWidth / 2;
 int origUIHeight = origHeight / 2;
 int uvIdx = uvWidth + uvHeight * (W / 2);

 int RGB[3] = { 0, 0, 255 };

 if (recurId == 0)
 {
  Line(yuv, beginX, beginY, width, height, RGB);
  return;
 }
 int movx = 0;
 int movy = 0;
 width + t > W ? movx = W : movx = width + t;
 height + t > H ? movy = H : movy = height + t;

 width - t < 0 ? movx = 0 : movx = width - t;
 height - t < H ? movy = 0 : movy = height - t;


 PCGRecur << <2, 4 >> >(beginX, beginY, width + t, height + t, yuv, tPCG, idx, recurIdx - 1, 0);
 //PCGRecur << <2, 1 >> >(beginX, beginY, width + t, height - t, yuv, tPCG, idx, recurIdx - 1, 0);
 //PCGRecur << <2, 1 >> >(beginX, beginY, width - t, height + t, yuv, tPCG, idx, recurIdx - 1, 0);
 //PCGRecur << <2, 1 >> >(beginX, beginY, width - t, height - t, yuv, tPCG, idx, recurIdx - 1, 0);
}
__global__ void PCG(Lab2VideoInfo &info, uint8_t * yuv, int tt)
{
 int t = tt;
 int idx = blockIdx.x * blockDim.x + threadIdx.x;

 int tPCG = t;
 //PCG2 << < 1, 1 >> > ();
 int width = idx % W;
 int height = idx / W;

 int uvWidth = width / 2;
 int uvHeight = height / 2;

 int uvIdx = uvWidth + uvHeight * (W / 2);
 int modT = t < 230 ? 240 - t : 0;
 int RGB[3] = { -1, -1, -1 };
 //int RGB[3] = { 255, 0, 0 };
 int *RGBtmp;
 for (int i = 1; i <= 1; i++)
 {

  if (i == 1) RGBtmp = PCGRecurOne(width, height, modT, i);

  for (int j = 0; j < 3; j++)
  {
   RGB[j] = RGBtmp[j];
  }

 }

 if (RGB[0] == -1 || RGB[1] == -1 || RGB[2] == -1) return;

 yuv[idx] = (int)RGBtoY(RGB[0], RGB[1], RGB[2]);
 yuv[W*H + uvIdx] = (int)RGBtoU(RGB[0], RGB[1], RGB[2]);
 yuv[W*H + W*H / 4 + uvIdx] = (int)RGBtoV(RGB[0], RGB[1], RGB[2]);


}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
 info.w = W;
 info.h = H;
 info.n_frame = NFRAME;
 // fps = 24/1 = 24
 info.fps_n = 24;
 info.fps_d = 1;


 tmpInfo = info;
};
__global__ void subTriangle(uint8_t *yuv, int n, float x1, float y1, float x2, float y2, float x3, float y3, int it)
{
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 int dist = 0;
 int maxDis = H;
 int RGB_Black[3] = { 255 - 100 * dist / maxDis, 0, 0 };
 //Draw the 3 sides as black lines
 Line(yuv, x1, y1, x2, y2, RGB_Black);
 Line(yuv, x1, y1, x3, y3, RGB_Black);
 Line(yuv, x2, y2, x3, y3, RGB_Black);

 //Calls itself 3 times with new corners, but only if the current number of recursions is smaller than the maximum depth
 if (n < it)
 {
  //Smaller triangle 1
  if (idx == 0)
  {
   subTriangle << <3, 1 >> >
    (
    yuv,
    n + 1, //Number of recursions for the next call increased with 1
    (x1 + x2) / 2 + (x2 - x3) / 2, //x coordinate of first corner
    (y1 + y2) / 2 + (y2 - y3) / 2, //y coordinate of first corner
    (x1 + x2) / 2 + (x1 - x3) / 2, //x coordinate of second corner
    (y1 + y2) / 2 + (y1 - y3) / 2, //y coordinate of second corner
    (x1 + x2) / 2, //x coordinate of third corner
    (y1 + y2) / 2, //y coordinate of third corner*
    it
    );
  }
  //Smaller triangle 2
  if (idx == 1)
  {
   subTriangle << <3, 1 >> >
    (
    yuv,
    n + 1, //Number of recursions for the next call increased with 1
    (x3 + x2) / 2 + (x2 - x1) / 2, //x coordinate of first corner
    (y3 + y2) / 2 + (y2 - y1) / 2, //y coordinate of first corner
    (x3 + x2) / 2 + (x3 - x1) / 2, //x coordinate of second corner
    (y3 + y2) / 2 + (y3 - y1) / 2, //y coordinate of second corner
    (x3 + x2) / 2, //x coordinate of third corner
    (y3 + y2) / 2,  //y coordinate of third corner
    it);
   //Smaller triangle 3
  }
  if (idx == 2)
  {
   subTriangle << <3, 1 >> >
    (
    yuv,
    n + 1, //Number of recursions for the next call increased with 1
    (x1 + x3) / 2 + (x3 - x2) / 2, //x coordinate of first corner
    (y1 + y3) / 2 + (y3 - y2) / 2, //y coordinate of first corner
    (x1 + x3) / 2 + (x1 - x2) / 2, //x coordinate of second corner
    (y1 + y3) / 2 + (y1 - y2) / 2, //y coordinate of second corner
    (x1 + x3) / 2, //x coordinate of third corner
    (y1 + y3) / 2,  //y coordinate of third corner
    it
    );
  }
 }
}
__global__ void drawSierpinski(uint8_t *yuv, float x1, float y1, float x2, float y2, float x3, float y3, int it)
{
 int RGB_Black[3] = {};
 //Draw the 3 sides of the triangle as black lines
 Line(yuv, x1, y1, x2, y2, RGB_Black);
 Line(yuv, x1, y1, x3, y3, RGB_Black);
 Line(yuv, x2, y2, x3, y3, RGB_Black);

 //Call the recursive function that'll draw all the rest. The 3 corners of it are always the centers of sides, so they're averages
 subTriangle << <3, 1 >> >
  (
  yuv,
  0, //This represents the first recursion
  (x1 + x2) / 2, //x coordinate of first corner
  (y1 + y2) / 2, //y coordinate of first corner
  (x1 + x3) / 2, //x coordinate of second corner
  (y1 + y3) / 2, //y coordinate of second corner
  (x2 + x3) / 2, //x coordinate of third corner
  (y2 + y3) / 2,  //y coordinate of third corner
  it);
}

//The recursive function that'll draw all the upside down triangles

void Lab2VideoGenerator::Generate(uint8_t *yuv) {
 Vector3D background_ColorRGB(255, 255, 255);
 //background
 //Y
 hipMemset(yuv, RGBtoY(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H);
 //U
 hipMemset(yuv + W*H, RGBtoU(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H / 4);
 //V
 hipMemset(yuv + W*H + W*H / 4, RGBtoV(background_ColorRGB.x, background_ColorRGB.y, background_ColorRGB.z), W*H / 4);

 int block_dim = H*W / W;
 int iterNum = 9;
 int t = iterNum * impl->t / NFRAME;
 //hipMemcpy(&t, &impl->t, sizeof(int), hipMemcpyHostToDevice);
 PCG << <block_dim, W >> >(tmpInfo, yuv, t);
 drawSierpinski << <1, 1 >> >(yuv, 10, H - 10, W - 10, H - 10, W / 2, 10, t); //Call the sierpinski function (works with any corners inside the screen

 ++(impl->t);
}